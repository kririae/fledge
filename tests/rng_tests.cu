#include <gtest/gtest.h>

#include "common/rng.h"
#include "fledge.h"
#include "gpu/gpu_rng.cuh"
#include "rng.hpp"

using namespace fledge;

TEST(RNG, Basic) {
  RandomGPU *rng_gpu_ptr;
  hipMallocManaged(&rng_gpu_ptr, sizeof(RandomGPU));
  new (rng_gpu_ptr) RandomGPU(114514);
}